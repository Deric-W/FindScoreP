#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <boost/program_options.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/driver_types.h>

namespace options = boost::program_options;

__device__ static unsigned long fibonacci(unsigned long element) {
    unsigned long a = 0;
    unsigned long b = 1;
    while (element > 0) {
        auto tmp = a + b;
        a = b;
        b = tmp;
        element--;
    }
    return a;
}

__global__ static void kernel(const unsigned int start, const unsigned int step, const unsigned int count, unsigned long* buffer) {
    for (unsigned int index = threadIdx.x; index < count; index += blockDim.x) {
        buffer[index] = fibonacci(start + index * step);
    }
}

static void calculate_elements(const unsigned int start, const unsigned int step, std::vector<unsigned long>* buffer) {
    unsigned long* device_buffer;
    hipMalloc(&device_buffer, sizeof(unsigned long) * buffer->size());
    kernel<<<1,256>>>(start, step, buffer->size(), device_buffer);
    hipMemcpy(buffer->data(), device_buffer, sizeof(unsigned long) * buffer->size(), hipMemcpyDeviceToHost);
    hipFree(device_buffer);
}

static void print_elements(const unsigned int start, const unsigned int step, std::vector<unsigned long>* buffer) {
    std::cout << "Calculated elements:\n";
    auto element = start;
    for (auto value: *buffer) {
        std::cout << element << ": " << value << "\n";
        element += step;
    }
}

int main(int argc, char** argv) {
    unsigned int start, step, count;

    options::options_description desc("Allowed options");
    desc.add_options()
        ("help,h", "display help message")
        ("start", options::wvalue<unsigned int>(&start)->required(), "starting index of the fibonacci sequence")
        ("step", options::value<unsigned int>(&step)->required(), "step size between the calculated values")
        ("count", options::value<unsigned int>(&count)->required(), "amount of elements to be calculated");
    options::positional_options_description positionals;
    positionals
        .add("start", 1)
        .add("step", 1)
        .add("count", 1);

    options::variables_map arguments;

    try {
        auto parsed = options::command_line_parser(argc, argv).options(desc).positional(positionals).style(options::command_line_style::unix_style).run();
        options::store(parsed, arguments);
        if (arguments.count("help")) {
            std::cout << "CUDA example which calculates elements of the fibonacci sequence" << "\n\n";
            std::cout << "Usage: fibonacci [OPTIONS] [start step count]" << "\n";
            std::cout << desc << "\n";
            return 0;
        }
        options::notify(arguments);
    } catch (options::error& e) {
        std::cerr << "Error while parsing arguments: " << e.what() << "\n";
        return 1;
    }

    std::vector<unsigned long> buffer(count);
    printf("Calculating elements with start: %u, step: %u and count: %u\n", start, step, count);
    calculate_elements(start, step, &buffer);
    print_elements(start, step, &buffer);
    return 0;
}